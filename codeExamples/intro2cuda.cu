
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>

using namespace std;

__global__ void add(int n, float* x, float* y){ //__global__ establishes device to managed by CUDA
    for (int i = 0; i < n; i++) {
        y[i] = x[i] + y[i];
    }
}

int main()
{
    int N = 1 << 20;

    float*x, *y;
    hipMallocManaged(&x, N * sizeof(float)); //allocate memory on GPU
    hipMallocManaged(&y, N * sizeof(float));

    for (int i = 0; i < N; i++) {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    add<<<1, 1>>>(N, x, y);

    hipDeviceSynchronize();

    float maxError = 0.0f;
    for (int i = 0; i < N; i++)
        maxError = fmax(maxError, fabs(y[i] - 3.0f));
    cout << "Max error:" << maxError << endl;

    delete[] x;
    delete[] y;

    return 0;

}
